#include "hip/hip_runtime.h"
#include <utility>
#include <cassert>

#ifdef COMPILE_FOR_CPU
#include <cmath>
#endif //COMPILE_FOR_CPU

#include "Face.cuh"
#include "../MapNode.cuh"
#include "Polyhedron.cuh"
#include "../../common.cuh"


__host__ __device__ SpacePoint calculate_normal(const SpacePoint *vertices)
{
    SpacePoint normal = (vertices[2] - vertices[0]) % (vertices[1] - vertices[0]);
    return normal / get_distance(normal, origin);
}


__host__ __device__ Face::Face(const SpacePoint *vertices, int n_of_vertices) :
        vertices(newalloc_and_copy(vertices, n_of_vertices)), n_of_vertices(n_of_vertices),
        normal(calculate_normal(vertices)), node(nullptr)
{

}

__host__ __device__ Face &Face::operator=(const Face &other)
{
    if(this != &other)
    {
        vertices = newalloc_and_copy(other.vertices, other.n_of_vertices);
        n_of_vertices = other.n_of_vertices;
        normal = other.normal;
        node = nullptr;
    }
    return *this;
}

__host__ __device__ Face::Face(const Face &other)
{
    *this = other;
}

__host__ __device__ Face &Face::operator=(Face &&other) noexcept
{
    if(this != &other)
    {
        swap(vertices, other.vertices);
        swap(n_of_vertices, other.n_of_vertices);
        swap(normal, other.normal);
        swap(node, other.node);
    }

    return *this;
}

__host__ __device__ Face::Face(Face &&other) noexcept
{
    vertices = nullptr;

    *this = std::move(other);
}

__host__ __device__ Face::Face()
{
    vertices = nullptr;
}

__host__ __device__ Face::~Face()
{
    delete[] vertices;
}


__host__ __device__ void Face::set_node(MapNode *node, Polyhedron *polyhedron)
{
    if(*this == *polyhedron->find_face_by_point(node->get_coordinates()))
    {
        this->node = node;
    }
}

__host__ __device__ MapNode *Face::get_node() const
{
    return node;
}

__host__ __device__ const SpacePoint *Face::get_vertices() const
{
    return vertices;
}

__host__ __device__ int Face::get_n_of_vertices() const
{
    return n_of_vertices;
}

__host__ __device__ SpacePoint Face::get_normal() const
{
    return normal;
}


__host__ __device__ bool operator==(const Face &a, const Face &b)
{
    if(a.n_of_vertices != b.n_of_vertices)
        return false;

    for(int i = 0; i < a.n_of_vertices; ++i)
    {
        if(a.vertices[i] != b.vertices[i])
            return false;
    }
    return true;
}

__host__ __device__ bool Face::contains_point(SpacePoint p)
{
    // For squares only!
    assert(n_of_vertices == 4);
    double face_area = get_distance(vertices[1], vertices[0]) * get_distance(vertices[2], vertices[1]);

    double face_area_with_point = 0;
    for(int i = 0; i < n_of_vertices; ++i)
    {
        SpacePoint this_vertex = vertices[i], next_vertex = vertices[(i + 1) % n_of_vertices];
        SpacePoint a = this_vertex - p, b = next_vertex - p;
        double triangle_area = 1. / 2 * sqrt((a * a) * (b * b) - (a * b) * (a * b));
        face_area_with_point += triangle_area;
    }

    // The face contains point if and only if the two areas are equal to each other
    return std::abs(face_area - face_area_with_point) < eps;
}


__host__ __device__ bool operator!=(const Face &a, const Face &b)
{
    return !(a == b);
}
